#include "head.cuh"


extern "C" void cudaMatrixMul(float *h_a, float *h_b, float *h_c, int M, int N, int K)
{
	int SIZE_A = M * K * sizeof(float);
	int SIZE_B = K * N * sizeof(float);
	int SIZE_C = M * N * sizeof(float);
	//device data
	float* device_A=0;
	float* device_B=0;
	float* device_C=0;
	hipSetDevice(0);
	//device memory
	CHECK(hipMalloc((void**)&device_A, SIZE_A));
	CHECK(hipMalloc((void**)&device_B, SIZE_B));
	CHECK(hipMalloc((void**)&device_C, SIZE_C));
	CHECK(hipMemcpy(device_A, h_a, SIZE_A, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(device_B, h_b, SIZE_B, hipMemcpyHostToDevice));

	//without share memory~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));
	CHECK(hipEventRecord(start, 0));
	hipEventQuery(start); //can not call by CHECK() used for WDDM mode GPU

	//noShareMemKernel(device_A, device_B, device_C, M, N, K);

	CHECK(hipDeviceSynchronize());
	//time end
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	float elapsedTime_cuda;
	CHECK(hipEventElapsedTime(&elapsedTime_cuda, start, stop));
	printf("no ShareMem Kernel time=%f ms\n\n", elapsedTime_cuda);
	CHECK(hipEventDestroy(start));
	CHECK(hipEventDestroy(stop));

	//double flops_cuda = 2.0 * (double)M * (double)N* (double)K;
	//double gigaFlops_cuda = (flops_cuda * 1.0e-9f) / (elapsedTime_cuda / 1000.0f);
	//printf("No Share Memory Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n", gigaFlops_cuda, elapsedTime_cuda, flops_cuda);


	//with shared memory~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
	hipEvent_t start1, stop1;
	CHECK(hipEventCreate(&start1));
	CHECK(hipEventCreate(&stop1));
	CHECK(hipEventRecord(start1, 0));
	hipEventQuery(start1);    //need for WDDM mode GPU =
	
	withSharedMemKernel(device_A, device_B, device_C, M, N, K);


	CHECK(hipDeviceSynchronize());
	CHECK(hipEventRecord(stop1, 0));
	CHECK(hipEventSynchronize(stop1));
	float elapsedTime_cuda_sharememory;
	CHECK(hipEventElapsedTime(&elapsedTime_cuda_sharememory, start1, stop1));
	printf("with SharedMem Kernel time=%f ms\n", elapsedTime_cuda_sharememory);
	CHECK(hipEventDestroy(start1));
	CHECK(hipEventDestroy(stop1));

	CHECK(hipMemcpy(h_c, device_C, SIZE_C, hipMemcpyDeviceToHost));
	
	
	//double flops_cuda_share = 2.0 * (double)M * (double)N* (double)K;
	//double gigaFlops_share = (flops_cuda_share * 1.0e-9f) / (elapsedTime_cuda_sharememory / 1000.0f);
	//printf("CUDA Share Memory Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n", gigaFlops_share, elapsedTime_cuda_sharememory, flops_cuda_share);
	//
	

	//free gpu memory
	CHECK(hipFree(device_A));
	CHECK(hipFree(device_B));
	CHECK(hipFree(device_C));

}

